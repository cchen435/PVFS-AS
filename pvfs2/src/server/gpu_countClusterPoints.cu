#include <stdio.h>
#include <stdlib.h>

//#include <cutil.h>
#include "kernel_countClusterPoints.cu"

#include "gpu_device.cuh"

void gpuCountClusterPoints(
			   int* d_membership,
			   int numDataPoints,
			   int numClusters,
			   int* d_newClusterHistogram
			   )
{
  /* Variables */
  int num_threads		= GPU_THREADS_PER_BLOCK;
  int num_blocks_x	= GPU_XBLOCKS_PER_GRID > (1 + ((numDataPoints - 1) / GPU_THREADS_PER_BLOCK)) ? (1 + ((numDataPoints - 1) / GPU_THREADS_PER_BLOCK)) : GPU_XBLOCKS_PER_GRID;

  /* Kernel Execution Parameters */
  dim3 grid_count(num_blocks_x, 1);
  dim3 block_count(num_threads, 1, 1);
  unsigned int sharedMem_count = sizeof(int) * numClusters * num_threads;

  /* During reduction each x-block takes a cluster */
  //int num_threads_reduce = GPU_THREADS_PER_BLOCK > num_blocks_x ? num_blocks_x : GPU_THREADS_PER_BLOCK;
  dim3 grid_countReduce(numClusters, 1);
  dim3 block_countReduce(num_threads, 1, 1);
  unsigned int sharedMem_countReduce = sizeof(int) * num_threads;

  //printf("Count Cluster Points ... [START]\n");
  //printf("	[INFO] GRID Config Count	: (%d, 1)\n", num_blocks_x);
  //printf("	[INFO] BLOCK Config Count	:	(%d, 1, 1)\n", num_threads);
  //printf("	[INFO] GRID Config Count Reduce	: (%d, 1)\n", numClusters);
  //printf("	[INFO] BLOCK Config Count Reduce:	(%d, 1, 1)\n", num_threads);

  /**/
  // Get an array with size num_blocks_x * numClusters
  int* d_clusterPointsCount = NULL;
  hipMalloc( (void**) &d_clusterPointsCount, sizeof(int) * num_blocks_x * numClusters);

  /* Kernel Invokation */
  switch(num_threads){
  case 512 :
    kernelCount<512> <<< grid_count, block_count, sharedMem_count>>> (d_membership, numDataPoints, numClusters, d_clusterPointsCount);
    kernelCountReduce<512> <<< grid_countReduce, block_countReduce, sharedMem_countReduce>>> (d_clusterPointsCount, num_blocks_x, d_newClusterHistogram);
    break;

  case 256 :
    kernelCount<256> <<< grid_count, block_count, sharedMem_count>>> (d_membership, numDataPoints, numClusters, d_clusterPointsCount);
    kernelCountReduce<256> <<< grid_countReduce, block_countReduce, sharedMem_countReduce>>> (d_clusterPointsCount, num_blocks_x, d_newClusterHistogram);
    break;

  case 128 :
    kernelCount<128> <<< grid_count, block_count, sharedMem_count>>> (d_membership, numDataPoints, numClusters, d_clusterPointsCount);
    kernelCountReduce<128> <<< grid_countReduce, block_countReduce, sharedMem_countReduce>>> (d_clusterPointsCount, num_blocks_x, d_newClusterHistogram);
    break;

  case 64 :
    kernelCount<64> <<< grid_count, block_count, sharedMem_count>>> (d_membership, numDataPoints, numClusters, d_clusterPointsCount);
    kernelCountReduce<64> <<< grid_countReduce, block_countReduce, sharedMem_countReduce>>> (d_clusterPointsCount, num_blocks_x, d_newClusterHistogram);
    break;

  case 32 :
    kernelCount<32> <<< grid_count, block_count, sharedMem_count>>> (d_membership, numDataPoints, numClusters, d_clusterPointsCount);
    kernelCountReduce<32> <<< grid_countReduce, block_countReduce, sharedMem_countReduce>>> (d_clusterPointsCount, num_blocks_x, d_newClusterHistogram);
    break;

  case 16 :
    kernelCount<16> <<< grid_count, block_count, sharedMem_count>>> (d_membership, numDataPoints, numClusters, d_clusterPointsCount);
    kernelCountReduce<16> <<< grid_countReduce, block_countReduce, sharedMem_countReduce>>> (d_clusterPointsCount, num_blocks_x, d_newClusterHistogram);
    break;

  case 8 :
    kernelCount<8> <<< grid_count, block_count, sharedMem_count>>> (d_membership, numDataPoints, numClusters, d_clusterPointsCount);
    kernelCountReduce<8> <<< grid_countReduce, block_countReduce, sharedMem_countReduce>>> (d_clusterPointsCount, num_blocks_x, d_newClusterHistogram);
    break;

  case 4 :
    kernelCount<4> <<< grid_count, block_count, sharedMem_count>>> (d_membership, numDataPoints, numClusters, d_clusterPointsCount);
    kernelCountReduce<4> <<< grid_countReduce, block_countReduce, sharedMem_countReduce>>> (d_clusterPointsCount, num_blocks_x, d_newClusterHistogram);
    break;

  case 2 :
    kernelCount<2> <<< grid_count, block_count, sharedMem_count>>> (d_membership, numDataPoints, numClusters, d_clusterPointsCount);
    kernelCountReduce<2> <<< grid_countReduce, block_countReduce, sharedMem_countReduce>>> (d_clusterPointsCount, num_blocks_x, d_newClusterHistogram);
    break;
  }
				
  //int* test_out = (int*)malloc(sizeof(int) * num_blocks_x * numClusters);
  //CUDA_SAFE_CALL( hipMemcpy(test_out, d_clusterPointsCount, sizeof(int) * num_blocks_x * numClusters, hipMemcpyDeviceToHost) );
  //printf("Test Out: %d\n", test_out[0]);
  //for(int i = 0; i < num_blocks_x; i++){
  //				for(int j = 0; j < numClusters; j++){
  //								printf("%d, ", test_out[num_blocks_x * i + j]);
  //				}
  //				printf("\n");
  //}
  //
  //test_out = (int*)malloc(sizeof(int) * numClusters);
  //CUDA_SAFE_CALL( hipMemcpy(test_out, d_newClusterHistogram, sizeof(int) * numClusters, hipMemcpyDeviceToHost) );
  //for(int i = 0; i < numClusters; i++) printf("%d, ", test_out[i]);
  //printf("\n");

  //int* abc = (int*)malloc(sizeof(int) * numDataPoints);
  //CUDA_SAFE_CALL( hipMemcpy(abc, d_membership, sizeof(int) * numDataPoints, hipMemcpyDeviceToHost) );
  //int sum = 0;
  //for(int i = 0; i < numDataPoints; i++) sum += (abc[i] + 1);
  //printf("Sum : %d\n", sum);

  //printf("Count Cluster Points ... [DONE]\n");

}
