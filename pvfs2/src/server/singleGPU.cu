#include "hip/hip_runtime.h"
/*
  For the Time being just use a single dimensional array.
*/
#include <stdlib.h>
#include <stdio.h>

#include <cutil_inline.h>
#include "gpu_kernels.cuh"

/* Used by computeDist.c */
float distanceEuclidean(float* refPoint, float* testPoint, int numDimensions){

  float sum = 0.0;
  int dim;

  for(dim = 0; dim < numDimensions; dim++)
    sum += (refPoint[dim] - testPoint[dim]) * (refPoint[dim] - testPoint[dim]);

  return sqrt(sum);
}

/* Used by computeError.c */
int errorAbsDifference(int newVal, int oldVal){

  return abs(newVal - oldVal);
}

int errorCheckEquality(int newVal, int oldVal){
  return newVal == oldVal ? 0 : 1;
}

float* d_dataPoints;
float* d_clusterCentres;
float* d_newClusterCentres;
int* d_membership;
int* d_oldMembership;
int* d_newMembership;
int* d_clusterHistogram;

extern "C" void initGPU(float* objects, int numDataPoints,
			int numDimensions, int numClusters)
{
#if DEBUG
  int i, j;
  float *h_newClusters = (float*)malloc(sizeof(float)*numClusters*numDimensions);
#endif

  /* Initialize the GPU device */
  CUT_DEVICE_INIT(1, "pvfs2-server");
  
  /* Print out Kmeans parameters */
  fprintf(stderr, "[INFO] Records: %d, Dimensions: %d, Clusters: %d\n", numDataPoints, numDimensions, numClusters);
#if DEBUG
  for(int i=0; i<numDimensions; i++)
    fprintf(stderr, "object[%d]=%f\n", i, objects[i]);
#endif

  /* Allocate memory on the device and copy the data */
  hipMalloc((void**) &d_dataPoints, sizeof(float)*numDataPoints*numDimensions);
  hipMalloc((void**) &d_clusterCentres, sizeof(float)*numClusters*numDimensions);
  hipMemset(d_clusterCentres, 0.0f, sizeof(float)*numClusters*numDimensions);
  hipMalloc((void**) &d_newClusterCentres, sizeof(float)*numClusters*numDimensions);
  hipMemset(d_newClusterCentres, 0.0f, sizeof(float)*numClusters*numDimensions);
#if DEBUG
  hipMemcpy(h_newClusters, d_newClusterCentres, sizeof(float)*numClusters*numDimensions, hipMemcpyDeviceToHost);
  for(i=0; i<numClusters; i++)
    for(j=0; j<numDimensions; j++)
      fprintf(stderr, "newClusters[%d][%d]=%f\n", i, j, h_newClusters[i*numDimensions+j]);
#endif
  hipMalloc((void**) &d_membership, sizeof(int)*numDataPoints);
  hipMalloc((void**) &d_oldMembership, sizeof(int)*numDataPoints);
  hipMalloc((void**) &d_newMembership, sizeof(int)*numDataPoints);
  hipMalloc((void**) &d_clusterHistogram, sizeof(int)*numClusters);

  /* Memory Initialization */
  hipMemcpy(d_dataPoints, objects, sizeof(float)*numDataPoints*numDimensions, hipMemcpyHostToDevice);

}

extern "C" void freeGPU(void)
{
  /* Free the device memory */
  hipFree(d_dataPoints);
  hipFree(d_clusterCentres);
  hipFree(d_newClusterCentres);
  hipFree(d_membership);
  hipFree(d_oldMembership);
  hipFree(d_newMembership);
  hipFree(d_clusterHistogram);

  hipDeviceReset();
}

extern "C" void singleGPU(float* objects, int numDataPoints, 
			  int numDimensions, float* clusters,
			  int numClusters, int *membership,
			  int *newClusterSize,
			  float* newClusters, float *error)
{
#if DEBUG
  int* h_clusterHistogram = (int*)malloc(sizeof(int)*numClusters);
  int i, j;
#endif

  hipMemcpy(d_clusterCentres, clusters, sizeof(float)*numClusters*numDimensions, hipMemcpyHostToDevice);
  hipMemcpy(d_newClusterCentres, newClusters, sizeof(float)*numClusters*numDimensions, hipMemcpyHostToDevice);
  hipMemcpy(d_oldMembership, membership, sizeof(int)*numDataPoints, hipMemcpyHostToDevice);

  /* Compute the distance */
  gpuComputeDistance(d_dataPoints, numDataPoints, numDimensions, d_clusterCentres, numClusters, d_newMembership, distanceEuclidean);

#if DEBUG
  hipMemcpy(membership, d_newMembership, sizeof(int)*numDataPoints, hipMemcpyDeviceToHost);

  for(i=0; i<20; i++)
    fprintf(stderr, "newMembership[%d]=%d\n", i, membership[i]);
#endif
  /* Compute the Histogram */
  gpuCountClusterPoints(d_newMembership, numDataPoints, numClusters, d_clusterHistogram);
#if DEBUG
  hipMemcpy(h_clusterHistogram, d_clusterHistogram, sizeof(int)*numClusters, hipMemcpyDeviceToHost);

  for(i=0; i<numClusters; i++)
    fprintf(stderr, "d_clusterHistogram[%d]=%d\n", i, h_clusterHistogram[i]);
#endif

  /* Update the Cluster Centres, using the d_clusterHistogram */
  //[MODIFY] I am for the time being updating the update cluster to give out the average value rather than summation
  gpuUpdateClusterCentres(d_newMembership, d_dataPoints, numDataPoints, numDimensions, numClusters, d_clusterHistogram, d_newClusterCentres);

  /* Compute the Error */
  /* result is copied back as a result of the call */
  gpuComputeError(d_newMembership, d_oldMembership, numDataPoints, error);
#if DEBUG
  fprintf(stderr, "error=%lf\n", *error);
  fprintf(stderr, "gpuComputeError ... [DONE]\n");
#endif

  /* Copy back the result to the host */
  /* newClusterCentres, newClusterSize, error */
  hipMemcpy(newClusters, d_newClusterCentres, sizeof(float)*numClusters*numDimensions, hipMemcpyDeviceToHost);
  hipMemcpy(newClusterSize, d_clusterHistogram, sizeof(int)*numClusters, hipMemcpyDeviceToHost);
#if DEBUG
  for(i=0; i<numClusters; i++)
    for(j=0; j<numDimensions; j++)
      fprintf(stderr, "newClusters[%d][%d]=%f\n", i, j, newClusters[i*numDimensions+j]);
#endif
  hipMemcpy(membership, d_newMembership, sizeof(int)*numDataPoints, hipMemcpyDeviceToHost);

  /* End Program Execution */
}
