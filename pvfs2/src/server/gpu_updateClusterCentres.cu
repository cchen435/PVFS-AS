#include <stdio.h>
#include <stdlib.h>

#include <cutil.h>
#include "kernel_updateClusterCentres.cu"

#include "gpu_device.cuh"

void gpuUpdateClusterCentres(
			     int* d_membership,
			     float* d_dataPoints,
			     int numDataPoints,
			     int numDimensions,
			     int numClusters,
			     int* d_clusterHistogram,
			     float* d_newClusterCentres
			     )
{
  /* Variables */
  int num_threads	= GPU_THREADS_PER_BLOCK;
  int num_blocks_x	= GPU_XBLOCKS_PER_GRID > (1 + ((numDataPoints - 1) / GPU_THREADS_PER_BLOCK)) ? (1 + ((numDataPoints - 1) / GPU_THREADS_PER_BLOCK)) : GPU_XBLOCKS_PER_GRID;
  int num_blocks_y	= numDimensions;

  /* Kernel Execution Parameters */
  dim3 grid_cluster(num_blocks_x, num_blocks_y); /* (128, 18) */
  dim3 block_cluster(num_threads, 1, 1); /* (128, 1, 1) */
  unsigned int sharedMem_cluster = sizeof(float) * numClusters * num_threads;

  /* During reduction each x-block takes a cluster and each y-block takes a dimension */
  //int num_threads_reduce = GPU_THREADS_PER_BLOCK > num_blocks_x ? num_blocks_x : GPU_THREADS_PER_BLOCK;
  dim3 grid_clusterReduce(numClusters, numDimensions);
  dim3 block_clusterReduce(num_threads, 1, 1);

  unsigned int sharedMem_clusterReduce = sizeof(float) * num_threads;

#if DEBUG
  fprintf(stderr, "Update Cluster Centres ... [START]\n");
  fprintf(stderr, "	[INFO] GRID Config Cluster	: (%d, %d)\n", num_blocks_x, num_blocks_y);
  fprintf(stderr, "	[INFO] BLOCK Config Cluster	:	(%d, 1, 1)\n", num_threads);
  fprintf(stderr, "	[INFO] GRID Config Cluster Reduce	: (%d, %d)\n", numClusters, numDimensions);
  fprintf(stderr, "	[INFO] BLOCK Config Cluster Reduce:	(%d, 1, 1)\n", num_threads);
#endif

  /**/
  // Get an array with size num_blocks_x * numClusters * numDimensions
  float* d_clusterCentresSum = NULL;
  float* h_clusterCentresSum = (float *)malloc(sizeof(float)*num_blocks_x*numClusters*numDimensions);
  hipMalloc( (void**) &d_clusterCentresSum, sizeof(float)*num_blocks_x*numClusters*numDimensions );
  hipMemset(d_clusterCentresSum, 0.0f, sizeof(float)*num_blocks_x*numClusters*numDimensions);

  /* Kernel Invokation */
  switch(num_threads){
  case 512 :
    kernelCluster<512> <<< grid_cluster, block_cluster, sharedMem_cluster>>> (d_dataPoints, numDataPoints, numDimensions, d_clusterCentresSum, numClusters, d_membership);
    kernelClusterReduce<512> <<< grid_clusterReduce, block_clusterReduce, sharedMem_clusterReduce>>> (d_clusterCentresSum, num_blocks_x, d_clusterHistogram, d_newClusterCentres);
    break;

  case 256 :
    kernelCluster<256> <<< grid_cluster, block_cluster, sharedMem_cluster>>> (d_dataPoints, numDataPoints, numDimensions, d_clusterCentresSum, numClusters, d_membership);
    kernelClusterReduce<256> <<< grid_clusterReduce, block_clusterReduce, sharedMem_clusterReduce>>> (d_clusterCentresSum, num_blocks_x, d_clusterHistogram, d_newClusterCentres);
    break;

  case 128 :
    kernelCluster<128> <<< grid_cluster, block_cluster, sharedMem_cluster>>> (d_dataPoints, numDataPoints, numDimensions, d_clusterCentresSum, numClusters, d_membership);
    kernelClusterReduce<128> <<< grid_clusterReduce, block_clusterReduce, sharedMem_clusterReduce>>> (d_clusterCentresSum, num_blocks_x, d_clusterHistogram, d_newClusterCentres);
    break;

  case 64 :
    kernelCluster<64> <<< grid_cluster, block_cluster, sharedMem_cluster>>> (d_dataPoints, numDataPoints, numDimensions, d_clusterCentresSum, numClusters, d_membership);
    kernelClusterReduce<64> <<< grid_clusterReduce, block_clusterReduce, sharedMem_clusterReduce>>> (d_clusterCentresSum, num_blocks_x, d_clusterHistogram, d_newClusterCentres);
    break;

  case 32 :
    kernelCluster<32> <<< grid_cluster, block_cluster, sharedMem_cluster>>> (d_dataPoints, numDataPoints, numDimensions, d_clusterCentresSum, numClusters, d_membership);
    kernelClusterReduce<32> <<< grid_clusterReduce, block_clusterReduce, sharedMem_clusterReduce>>> (d_clusterCentresSum, num_blocks_x, d_clusterHistogram, d_newClusterCentres);
    break;

  case 16 :
    kernelCluster<16> <<< grid_cluster, block_cluster, sharedMem_cluster>>> (d_dataPoints, numDataPoints, numDimensions, d_clusterCentresSum, numClusters, d_membership);
    kernelClusterReduce<16> <<< grid_clusterReduce, block_clusterReduce, sharedMem_clusterReduce>>> (d_clusterCentresSum, num_blocks_x, d_clusterHistogram, d_newClusterCentres);
    break;

  case 8 :
    kernelCluster<8> <<< grid_cluster, block_cluster, sharedMem_cluster>>> (d_dataPoints, numDataPoints, numDimensions, d_clusterCentresSum, numClusters, d_membership);
    kernelClusterReduce<8> <<< grid_clusterReduce, block_clusterReduce, sharedMem_clusterReduce>>> (d_clusterCentresSum, num_blocks_x, d_clusterHistogram, d_newClusterCentres);
    break;

  case 4 :
    kernelCluster<4> <<< grid_cluster, block_cluster, sharedMem_cluster>>> (d_dataPoints, numDataPoints, numDimensions, d_clusterCentresSum, numClusters, d_membership);
    kernelClusterReduce<4> <<< grid_clusterReduce, block_clusterReduce, sharedMem_clusterReduce>>> (d_clusterCentresSum, num_blocks_x, d_clusterHistogram, d_newClusterCentres);
    break;

  case 2 :
    kernelCluster<2> <<< grid_cluster, block_cluster, sharedMem_cluster>>> (d_dataPoints, numDataPoints, numDimensions, d_clusterCentresSum, numClusters, d_membership);
    kernelClusterReduce<2> <<< grid_clusterReduce, block_clusterReduce, sharedMem_clusterReduce>>> (d_clusterCentresSum, num_blocks_x, d_clusterHistogram, d_newClusterCentres);
    break;

  }

#if DEBUG				
  float* test_out = (float*)malloc(sizeof(float) * num_blocks_x * numClusters * numDimensions);
  hipMemcpy(test_out, d_clusterCentresSum, sizeof(float) * num_blocks_x * numClusters * numDimensions, hipMemcpyDeviceToHost);

  fprintf(stderr, "Test Out : ");
  for(int i = 0; i < numDimensions; i++) fprintf(stderr, "%f, ", test_out[i]);
  fprintf(stderr, "\n");
#endif	

}
