#include <stdio.h>
#include <stdlib.h>

//#include <cutil.h>
#include "kernel_computeError.cu"

#include "gpu_device.cuh"

void gpuComputeError(int* d_newMembership,
		     int* d_oldMembership,
		     int numDataPoints,
		     float* error)
{
  /* Variables */
  int num_threads	= GPU_THREADS_PER_BLOCK;
  int num_blocks_x	= GPU_XBLOCKS_PER_GRID > (1 + ((numDataPoints - 1) / GPU_THREADS_PER_BLOCK)) ? (1 + ((numDataPoints - 1) / GPU_THREADS_PER_BLOCK)) : GPU_XBLOCKS_PER_GRID;

  /* Kernel Execution parameters */
  dim3 grid_partialError(num_blocks_x, 1);
  dim3 block_partialError(num_threads, 1, 1);
  unsigned int sharedMem_partialError = sizeof(int) * num_threads;

  dim3 grid_error(1, 1);
  dim3 block_error(num_threads, 1, 1);
  unsigned int sharedMem_error = sizeof(int) * num_threads;

  //printf("Error Computation ... [START]\n");
  //printf("	[INFO] GRID Config Error	: (%d, 1)\n", num_blocks_x);
  //printf("	[INFO] BLOCK Config Error	:	(%d, 1, 1)\n", num_threads);
  //printf("	[INFO] GRID Config Error Reduce	: (1, 1)\n");
  //printf("	[INFO] BLOCK Config Error Reduce:	(%d, 1, 1)\n", num_threads);

  float* d_error = NULL;
  float* d_partialError = NULL;
  hipMalloc( (void**) &d_error, sizeof(float) );
  hipMalloc( (void**) &d_partialError, sizeof(float) * num_blocks_x);

  /* Kernel Execution */
  switch(num_threads)
    {
    case 512: 
      kernelError<512> <<< grid_partialError, block_partialError, sharedMem_partialError >>> (d_newMembership, d_oldMembership, numDataPoints, d_partialError);
      kernelErrorReduce<512> <<< grid_error, block_error, sharedMem_error >>> (d_partialError, num_blocks_x, d_error);
      break;

    case 256: 
      kernelError<256> <<< grid_partialError, block_partialError, sharedMem_partialError >>> (d_newMembership, d_oldMembership, numDataPoints, d_partialError);
      kernelErrorReduce<256> <<< grid_error, block_error, sharedMem_error >>> (d_partialError, num_blocks_x, d_error);
      break;

    case 128: 
      kernelError<128> <<< grid_partialError, block_partialError, sharedMem_partialError >>> (d_newMembership, d_oldMembership, numDataPoints, d_partialError);
      kernelErrorReduce<128> <<< grid_error, block_error, sharedMem_error >>> (d_partialError, num_blocks_x, d_error);
      break;

    case 64: 
      kernelError<64> <<< grid_partialError, block_partialError, sharedMem_partialError >>> (d_newMembership, d_oldMembership, numDataPoints, d_partialError);
      kernelErrorReduce<64> <<< grid_error, block_error, sharedMem_error >>> (d_partialError, num_blocks_x, d_error);
      break;

    case 32: 
      kernelError<32> <<< grid_partialError, block_partialError, sharedMem_partialError >>> (d_newMembership, d_oldMembership, numDataPoints, d_partialError);
      kernelErrorReduce<32> <<< grid_error, block_error, sharedMem_error >>> (d_partialError, num_blocks_x, d_error);
      break;

    case 16: 
      kernelError<16> <<< grid_partialError, block_partialError, sharedMem_partialError >>> (d_newMembership, d_oldMembership, numDataPoints, d_partialError);
      kernelErrorReduce<16> <<< grid_error, block_error, sharedMem_error >>> (d_partialError, num_blocks_x, d_error);
      break;

    case 8: 
      kernelError<8> <<< grid_partialError, block_partialError, sharedMem_partialError >>> (d_newMembership, d_oldMembership, numDataPoints, d_partialError);
      kernelErrorReduce<8> <<< grid_error, block_error, sharedMem_error >>> (d_partialError, num_blocks_x, d_error);
      break;

    case 4: 
      kernelError<4> <<< grid_partialError, block_partialError, sharedMem_partialError >>> (d_newMembership, d_oldMembership, numDataPoints, d_partialError);
      kernelErrorReduce<4> <<< grid_error, block_error, sharedMem_error >>> (d_partialError, num_blocks_x, d_error);
      break;

    case 2: 
      kernelError<2> <<< grid_partialError, block_partialError, sharedMem_partialError >>> (d_newMembership, d_oldMembership, numDataPoints, d_partialError);
      kernelErrorReduce<2> <<< grid_error, block_error, sharedMem_error >>> (d_partialError, num_blocks_x, d_error);
      break;

    }

  hipMemcpy(error, d_error, sizeof(float), hipMemcpyDeviceToHost);
				
  //printf("Error Computation ... [DONE]\n");
}
