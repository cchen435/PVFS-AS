#include <stdio.h>

#include <cutil.h>

void gpuInitializeMemory(
			 float** h_dataPoints,
			 float** h_clusterCentres,
			 int* h_membership,
			 int numDataPoints,
			 int numClusters,
			 int numDimensions,
			 float* d_dataPoints,
			 float* d_clusterCentres,
			 int* d_membership,
			 size_t d_pitch_dataPoints,
			 size_t d_pitch_clusterCentres
			 )
{

  /* Define the variables */
  int widthBytes_dataPoints			= sizeof(float) * numDimensions;
  int widthBytes_clusterCentres	= sizeof(float) * numDimensions;
  int size_membership						= sizeof(int) * numDataPoints;

  /* Copy 2D host memory to device memory*/
  CUDA_SAFE_CALL( hipMemcpy2D(d_dataPoints, d_pitch_dataPoints, h_dataPoints, widthBytes_dataPoints, widthBytes_dataPoints, numDataPoints, hipMemcpyHostToDevice) );
  CUDA_SAFE_CALL( hipMemcpy2D(d_clusterCentres, d_pitch_clusterCentres, h_clusterCentres, widthBytes_clusterCentres, widthBytes_clusterCentres, numClusters, hipMemcpyHostToDevice) );

  /* Copy 1D host memory to the device memory*/
  CUDA_SAFE_CALL( hipMalloc(d_membership, h_membership, size_membership, hipMemcpyHostToDevice) );
}
