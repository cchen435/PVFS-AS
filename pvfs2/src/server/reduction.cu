#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.  This source code is a "commercial item" as
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer software" and "commercial computer software
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 */

/*
  Parallel reduction

  This sample shows how to perform a reduction operation on an array of values
  to produce a single value.

  Reductions are a very common computation in parallel algorithms.  Any time
  an array of values needs to be reduced to a single value using a binary 
  associative operator, a reduction can be used.  Example applications include
  statistics computaions such as mean and standard deviation, and image 
  processing applications such as finding the total luminance of an
  image.

  This code performs sum reductions, but any associative operator such as
  min() or max() could also be used.

  It assumes the input size is a power of 2.

  COMMAND LINE ARGUMENTS

  "--shmoo":         Test performance for 1 to 32M elements with each of the 7 different kernels
  "--n=<N>":         Specify the number of elements to reduce (default 1048576)
  "--threads=<N>":   Specify the number of threads per block (default 128)
  "--kernel=<N>":    Specify which kernel to run (0-6, default 6)
  "--maxblocks=<N>": Specify the maximum number of thread blocks to launch (kernel 6 only, default 64)
  "--cpufinal":      Read back the per-block results and do final sum of block sums on CPU (default false)
  "--cputhresh=<N>": The threshold of number of blocks sums below which to perform a CPU final reduction (default 1)
    
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil_inline.h>

#include <reduction_kernel.cu>
int operation = 1;

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

extern "C"
float reduceGold(float *data, int size, int operation);

////////////////////////////////////////////////////////////////////////////////
// Wrapper function for kernel launch
////////////////////////////////////////////////////////////////////////////////
void reduce(int size, int threads, int blocks, int whichKernel, float *d_idata, float *d_odata, int value)
{
  dim3 dimBlock(threads, 1, 1);
  dim3 dimGrid(blocks, 1, 1);
  int smemSize = threads * sizeof(int);

  // choose which of the optimized versions of reduction to launch
  switch (whichKernel)
    {
    case 0:
      reduce0<<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata);
      break;
    case 1:
      reduce1<<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata);
      break;
    case 2:
      reduce2<<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata);
      break;
    case 3:
      reduce3<<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata);
      break;
    case 4:
      reduce4<<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata);
      break;
    case 5:
    default:
      switch (threads)
        {
        case 512:
	  reduce5<512><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case 256:
	  reduce5<256><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case 128:
	  reduce5<128><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case 64:
	  reduce5< 64><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case 32:
	  reduce5< 32><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case 16:
	  reduce5< 16><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case  8:
	  reduce5<  8><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case  4:
	  reduce5<  4><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case  2:
	  reduce5<  2><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        case  1:
	  reduce5<  1><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata); break;
        }
      break;       
    case 6:
      switch (threads)
        {
        case 512:
	  reduce6<512><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, operation, value); break;
        case 256:
	  reduce6<256><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, operation, value); break;
        case 128:
	  reduce6<128><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, operation, value); break;
        case 64:
	  reduce6< 64><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, operation, value); break;
        case 32:
	  reduce6< 32><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, operation, value); break;
        case 16:
	  reduce6< 16><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, operation, value); break;
        case  8:
	  reduce6<  8><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, operation, value); break;
        case  4:
	  reduce6<  4><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, operation, value); break;
        case  2:
	  reduce6<  2><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, operation, value); break;
        case  1:
	  reduce6<  1><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size, operation, value); break;
        }
      break;       
    }
}

////////////////////////////////////////////////////////////////////////////////
// Compute the number of threads and blocks to use for the given reduction kernel
// For the kernels >= 3, we set threads / block to the minimum of maxThreads and
// n/2. For kernels < 3, we set to the minimum of maxThreads and n.  For kernel 
// 6, we observe the maximum specified number of blocks, because each thread in 
// that kernel can process a variable number of elements.
////////////////////////////////////////////////////////////////////////////////
void getNumBlocksAndThreads(int whichKernel, int n, int maxBlocks, int maxThreads, int &blocks, int &threads)
{
  if (whichKernel < 3)
    {
      threads = (n < maxThreads) ? n : maxThreads;
      blocks = n / threads;
    }
  else
    {
      if (n == 1) 
	threads = 1;
      else
	threads = (n < maxThreads*2) ? n / 2 : maxThreads;
      blocks = n / (threads * 2);

      if (whichKernel == 6)
	blocks = min(maxBlocks, blocks);
    }
}

////////////////////////////////////////////////////////////////////////////////
// This function performs a reduction of the input data multiple times and 
// measures the average reduction time.
////////////////////////////////////////////////////////////////////////////////
float benchmarkReduce(int  n, 
		      int  numThreads,
		      int  numBlocks,
		      int  maxThreads,
		      int  maxBlocks,
		      int  whichKernel, 
		      int  testIterations,
		      bool cpuFinalReduction,
		      int  cpuFinalThreshold,
		      unsigned int timer,
		      float* h_odata,
		      float* d_idata, 
		      float* d_odata)
{
  float gpu_result = 0.0f;
  bool needReadBack = true;
  float value=0.0f;

  for (int i = 0; i < testIterations; ++i)
    {
      gpu_result = 0.0f;

      hipDeviceSynchronize();
      //CUT_SAFE_CALL( cutStartTimer( timer));

      // execute the kernel
      reduce(n, numThreads, numBlocks, whichKernel, d_idata, d_odata, value);

      // check if kernel execution generated an error
      CUT_CHECK_ERROR("Kernel execution failed");

      if (cpuFinalReduction)
        {
	  // sum partial sums from each block on CPU        
	  // copy result from device to host
	  CUDA_SAFE_CALL( hipMemcpy( h_odata, d_odata, numBlocks*sizeof(float), hipMemcpyDeviceToHost) );

	  for(int i=0; i<numBlocks; i++) 
            {
	      gpu_result += h_odata[i];
            }

	  needReadBack = false;
        }
      else
        {
	  // sum partial block sums on GPU
	  int s=numBlocks;
	  int kernel = (whichKernel == 6) ? 5 : whichKernel;
	  while(s > cpuFinalThreshold) 
            {
	      int threads = 0, blocks = 0;
	      getNumBlocksAndThreads(kernel, s, maxBlocks, maxThreads, blocks, threads);
	      reduce(s, threads, blocks, kernel, d_odata, d_odata, value);
	      if (kernel < 3)
		s = s / threads;
	      else
		s = s / (threads*2);
            }
            
	  if (s > 1)
            {
	      // copy result from device to host
	      CUDA_SAFE_CALL( hipMemcpy( h_odata, d_odata, s * sizeof(float), hipMemcpyDeviceToHost) );

	      for(int i=0; i < s; i++) 
                {
		  gpu_result += h_odata[i];
                }

	      needReadBack = false;
            }
        }

      hipDeviceSynchronize();
      CUT_SAFE_CALL( cutStopTimer(timer) );      
    }

  if (needReadBack)
    {
      // copy final sum from device to host
      CUDA_SAFE_CALL( hipMemcpy( &gpu_result, d_odata, sizeof(float), hipMemcpyDeviceToHost) );
    }

  return gpu_result;
}

#if 0
////////////////////////////////////////////////////////////////////////////////
// This function calls benchmarkReduce multple times for a range of array sizes
// and prints a report in CSV (comma-separated value) format that can be used for
// generating a "shmoo" plot showing the performance for each kernel variation
// over a wide range of input sizes.
////////////////////////////////////////////////////////////////////////////////
void shmoo(int minN, int maxN, int maxThreads, int maxBlocks)
{ 
  // create random input data on CPU
  unsigned int bytes = maxN * sizeof(int);
  float value=0.0f;

  float *h_idata = (float *) malloc(bytes);

  for(int i = 0; i < maxN; i++) {
    h_idata[i] = rand() & 0xff;
  }

  int maxNumBlocks = maxN / maxThreads;

  // allocate mem for the result on host side
  float* h_odata = (float*) malloc(maxNumBlocks*sizeof(float));

  // allocate device memory and data
  float* d_idata = NULL;
  float* d_odata = NULL;

  CUDA_SAFE_CALL( hipMalloc((void**) &d_idata, bytes) );
  CUDA_SAFE_CALL( hipMalloc((void**) &d_odata, maxNumBlocks*sizeof(float)) );

  // copy data directly to device memory
  CUDA_SAFE_CALL( hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice) );
  CUDA_SAFE_CALL( hipMemcpy(d_odata, h_idata, maxNumBlocks*sizeof(float), hipMemcpyHostToDevice) );

  // warm-up
#ifndef __DEVICE_EMULATION__
  for (int kernel = 0; kernel < 7; kernel++)
    {
      reduce(maxN, maxThreads, maxNumBlocks, kernel, d_idata, d_odata, value);
    }
  int testIterations = 100;
#else
  int testIterations = 1;
#endif

  unsigned int timer = 0;
  //CUT_SAFE_CALL( cutCreateTimer( &timer));
    
  // print headers
  printf("Time in milliseconds for various numbers of elements for each kernel\n");
  printf("\n\n");
  printf("Kernel");
  for (int i = minN; i <= maxN; i *= 2)
    {
      printf(", %d", i);
    }
   
  for (int kernel = 0; kernel < 7; kernel++)
    {
      printf("\n");
      printf("%d", kernel);
      for (int i = minN; i <= maxN; i *= 2)
        {
	  //cutResetTimer(timer);
	  int numBlocks = 0;
	  int numThreads = 0;
	  getNumBlocksAndThreads(kernel, i, maxBlocks, maxThreads, numBlocks, numThreads);
            
            
	  benchmarkReduce(i, numThreads, numBlocks, maxThreads, maxBlocks, kernel, 
			  testIterations, false, 1, timer, h_odata, d_idata, d_odata);

	  float reduceTime = cutGetAverageTimerValue(timer);
	  printf(", %f", reduceTime);
        }
        
    }

  // cleanup
  CUT_SAFE_CALL(cutDeleteTimer(timer));
  free(h_idata);
  free(h_odata);

  CUDA_SAFE_CALL(hipFree(d_idata));
  CUDA_SAFE_CALL(hipFree(d_odata));    
}
#endif

////////////////////////////////////////////////////////////////////////////////
// The main function which runs the reduction test.
////////////////////////////////////////////////////////////////////////////////
void
//reductionGPU( int argc, char** argv) 
reductionGPU(int size, int operation)
{
  CUT_DEVICE_INIT(1, "pvfs2-server");

  //int size = 1<<20;    // number of elements to reduce
  int maxThreads = 128;  // number of threads per block
  int whichKernel = 6;
  int maxBlocks = 64;
  bool cpuFinalReduction = false;
  int cpuFinalThreshold = 1;
  float value=0.0f;

  //cutGetCmdLineArgumenti( argc, (const char**) argv, "operation", &operation);
  //cutGetCmdLineArgumenti( argc, (const char**) argv, "n", &size);
  //cutGetCmdLineArgumenti( argc, (const char**) argv, "threads", &maxThreads);
  //cutGetCmdLineArgumenti( argc, (const char**) argv, "kernel", &whichKernel);
  //cutGetCmdLineArgumenti( argc, (const char**) argv, "maxblocks", &maxBlocks);
  printf("%d elements\n", size);
  printf("%d threads (max)\n", maxThreads);

  //cpuFinalReduction = cutCheckCmdLineFlag( argc, (const char**) argv, "cpufinal");
  //cutGetCmdLineArgumenti( argc, (const char**) argv, "cputhresh", &cpuFinalThreshold);

  //bool runShmoo = cutCheckCmdLineFlag(argc, (const char**) argv, "shmoo");
  bool runShmoo;

  if (runShmoo)
    {
      //shmoo(1, 33554432, maxThreads, maxBlocks);
    }
  else
    {
      // create random input data on CPU
      unsigned int bytes = size * sizeof(float);

      float *h_idata = (float *) malloc(bytes);

      for(int i=0; i<size; i++) {
	h_idata[i] = size-i;//i+1;//rand() & 1;
      }

      int numBlocks = 0;
      int numThreads = 0;
      getNumBlocksAndThreads(whichKernel, size, maxBlocks, maxThreads, numBlocks, numThreads);
      if (numBlocks == 1) cpuFinalThreshold = 1;

      // allocate mem for the result on host side
      float* h_odata = (float*) malloc(numBlocks*sizeof(float));

      printf("%d blocks\n", numBlocks);

      // allocate device memory and data
      float* d_idata = NULL;
      float* d_odata = NULL;

      CUDA_SAFE_CALL( hipMalloc((void**) &d_idata, bytes) );
      CUDA_SAFE_CALL( hipMalloc((void**) &d_odata, numBlocks*sizeof(float)) );

      // copy data directly to device memory
      CUDA_SAFE_CALL( hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice) );
      CUDA_SAFE_CALL( hipMemcpy(d_odata, h_idata, numBlocks*sizeof(float), hipMemcpyHostToDevice) );

      // warm-up
#ifndef __DEVICE_EMULATION__
      reduce(size, numThreads, numBlocks, whichKernel, d_idata, d_odata, value);
      int testIterations = 100;
#else
      int testIterations = 1;
#endif

      unsigned int timer = 0;
      CUT_SAFE_CALL( cutCreateTimer( &timer));
      unsigned int timerCPU = 0;
      CUT_SAFE_CALL( cutCreateTimer( &timerCPU));
        
      float gpu_result = 0.0f;

      hipDeviceSynchronize();
      CUT_SAFE_CALL( cutStartTimer( timer));
      switch (operation)
	{
	case 4:
	  operation=1;
	  reduce(size, numThreads, numBlocks, 6,d_idata,d_odata, value);
	  // copy final sum from device to host
	  CUDA_SAFE_CALL( hipMemcpy( h_odata, d_odata, numBlocks*sizeof(float), hipMemcpyDeviceToHost) );
	  for(int i=0; i<numBlocks; i++) 
	    {
	      gpu_result += h_odata[i];
	    }
	  gpu_result/=size;
	  operation=4;
	  break;
	case 5:	
	  operation=1;
	  reduce(size, numThreads, numBlocks, 6,d_idata,d_odata, value);
	  // copy final sum from device to host
	  CUDA_SAFE_CALL( hipMemcpy( h_odata, d_odata, numBlocks*sizeof(float), hipMemcpyDeviceToHost) );
	  for(int i=0; i<numBlocks; i++) 
	    {
	      gpu_result += h_odata[i];
	    }
	  //averagi hesapla
	  gpu_result/=size;
	  operation=5;
	  reduce(size, numThreads, numBlocks, 6,d_idata,d_odata, gpu_result);
	  // copy final sum from device to host
	  CUDA_SAFE_CALL( hipMemcpy( h_odata, d_odata, numBlocks*sizeof(float), hipMemcpyDeviceToHost) );
	  gpu_result=0;
	  for(int i=0; i<numBlocks; i++) 
	    {
	      gpu_result += h_odata[i];
	    }			
	  gpu_result/=size;
	  break;
	case 6:
	  operation=1;
	  reduce(size, numThreads, numBlocks, 6,d_idata,d_odata, value);
	  // copy final sum from device to host
	  CUDA_SAFE_CALL( hipMemcpy( h_odata, d_odata, numBlocks*sizeof(float), hipMemcpyDeviceToHost) );
	  for(int i=0; i<numBlocks; i++)
	    {
	      gpu_result += h_odata[i];
	    }
	  //averagi hesapla
	  gpu_result/=size;
	  operation=5;
	  reduce(size, numThreads, numBlocks, 6,d_idata,d_odata, gpu_result);
	  // copy final sum from device to host
	  CUDA_SAFE_CALL( hipMemcpy( h_odata, d_odata, numBlocks*sizeof(float), hipMemcpyDeviceToHost) );
	  gpu_result=0;
	  for(int i=0; i<numBlocks; i++)
	    {
	      gpu_result += h_odata[i];
	    }
	  gpu_result/=size;
	  gpu_result=sqrt(gpu_result);
	  break;
	case 1: 
	  reduce(size, numThreads, numBlocks, 6,d_idata,d_odata, value);
	  // copy final sum from device to host
	  CUDA_SAFE_CALL( hipMemcpy( h_odata, d_odata, numBlocks*sizeof(float), hipMemcpyDeviceToHost) );
	  for(int i=0; i<numBlocks; i++) 
	    {
	      // printf("%d h_odatap[%d]\n",h_odata[i],i);
	      gpu_result += h_odata[i];
	    }
	  break;
	case 2: 
	  reduce(size, numThreads, numBlocks, 6,d_idata,d_odata, value);
	  // copy final sum from device to host
	  CUDA_SAFE_CALL( hipMemcpy( h_odata, d_odata, numBlocks*sizeof(float), hipMemcpyDeviceToHost) );
	  gpu_result=h_odata[0];
	  //printf("%d h_odatap[0]\n",gpu_result);
	  for(int i=1; i<numBlocks; i++) 
	    {
	      // printf("%d h_odatap[%d]\n",h_odata[i],i);
	      if (gpu_result > h_odata[i]) gpu_result = h_odata[i];
	    }
	  break;
	case 3: 
	  reduce(size, numThreads, numBlocks, 6,d_idata,d_odata, value);
	  // copy final sum from device to host
	  CUDA_SAFE_CALL( hipMemcpy( h_odata, d_odata, numBlocks*sizeof(float), hipMemcpyDeviceToHost) );
	  gpu_result=h_odata[0];
	  //printf("%d h_odatap[0]\n",gpu_result);
	  for(int i=1; i<numBlocks; i++) 
	    {
	      // 				printf("%d h_odatap[%d]\n",h_odata[i],i);
	      if (gpu_result < h_odata[i]) gpu_result = h_odata[i];
	    }
	  break;
	}
      hipDeviceSynchronize();
      //CUT_SAFE_CALL( cutStopTimer(timer) );

      //float reduceTime = cutGetAverageTimerValue(timer);
      //printf("Average time: %f ms\n", reduceTime);
      //printf("Bandwidth:    %f GB/s\n\n", (size * sizeof(float)) / (reduceTime * 1.0e6));

      //printf("GPU result = %f\n", gpu_result);


      //CUT_SAFE_CALL( cutStartTimer( timerCPU));
      // compute reference solution
      //long cpu_result = reduceGold(h_idata, size,operation);
      //printf("CPU result = %f\n", cpu_result);
      //CUT_SAFE_CALL( cutStopTimer(timerCPU) );
      //float reduceTimeCPU = cutGetAverageTimerValue(timerCPU);
      //printf("Average time: %f ms\n", reduceTimeCPU);

      //printf("TEST %s\n", (gpu_result == cpu_result) ? "PASSED" : "FAILED");

      // cleanup
      CUT_SAFE_CALL( cutDeleteTimer(timer) );
      free(h_idata);
      free(h_odata);

      CUDA_SAFE_CALL(hipFree(d_idata));
      CUDA_SAFE_CALL(hipFree(d_odata));
    }
}
