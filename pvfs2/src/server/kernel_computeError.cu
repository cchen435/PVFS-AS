
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

template <unsigned int blockSize>
__global__ void kernelError(
			    int* d_newMembership,
			    int* d_oldMembership,
			    int numDataPoints,
			    float* d_partialError
			    )
{
  /* Shared Memory size = blockSize */
  extern __shared__ float s_partialError[];

  /* Threads IDs*/
  unsigned int tidx = threadIdx.x;
  unsigned int block_col = blockIdx.x;
  unsigned int gridSize = gridDim.x * blockSize;

  /* Initialize the shared memory */
  s_partialError[tidx] = 0;

  /* Compute the error */
  unsigned int ep = (blockSize * block_col) + tidx;
  while(ep < numDataPoints){
    // You can use a ternary operator too!!!
    s_partialError[tidx] += (d_newMembership[ep] == d_oldMembership[ep] ? 0 : 1);
    //s_partialError[tidx] += abs(d_newMembership[ep] - d_oldMembership[ep]);
    ep += gridSize;
  }
  __syncthreads();

  /* Do the partial reduction in the shared memory */
  if(blockSize >= 512){
    if(tidx < 256) s_partialError[tidx] += s_partialError[tidx + 256];
    __syncthreads();
  }
				
  if(blockSize >= 256){
    if(tidx < 128) s_partialError[tidx] += s_partialError[tidx + 128];
    __syncthreads();
  }
				
  if(blockSize >= 128){
    if(tidx < 64) s_partialError[tidx] += s_partialError[tidx + 64];
    __syncthreads();
  }

  if(tidx < 32){
    if(blockSize >= 64) s_partialError[tidx] += s_partialError[tidx + 32];
    if(blockSize >= 32) s_partialError[tidx] += s_partialError[tidx + 16];
    if(blockSize >= 16) s_partialError[tidx] += s_partialError[tidx + 8];
    if(blockSize >= 8) s_partialError[tidx] += s_partialError[tidx + 4];
    if(blockSize >= 4) s_partialError[tidx] += s_partialError[tidx + 2];
    if(blockSize >= 2) s_partialError[tidx] += s_partialError[tidx + 1];
  }

  /* Write the results to a remperoray array for further reduction */
  if(tidx == 0){
    d_partialError[block_col] = s_partialError[0];
  }
}

template <unsigned int blockSize>
__global__ void kernelErrorReduce(
				  float* d_partialError,
				  int numPartialErrorPoints,
				  float* d_error
				  )
{
  extern __shared__ float s_error[];

  /* Thread IDs*/
  unsigned int tidx = threadIdx.x;
				
  /* Initialize the shared data */
  s_error[tidx] = 0;

  /* Copy data to the shared memory */
  for(unsigned int i = tidx; i < numPartialErrorPoints; i += blockSize)
    s_error[tidx] += d_partialError[i];

  __syncthreads();

  /* Do the complete reduction in the shared memory */
  if(blockSize >= 512){
    if(tidx < 256) s_error[tidx] += s_error[tidx + 256];
    __syncthreads();
  }
				
  if(blockSize >= 256){
    if(tidx < 128) s_error[tidx] += s_error[tidx + 128];
    __syncthreads();
  }
				
  if(blockSize >= 128){
    if(tidx < 64) s_error[tidx] += s_error[tidx + 64];
    __syncthreads();
  }

  if(tidx < 32){
    if(blockSize >= 64) s_error[tidx] += s_error[tidx + 32];
    if(blockSize >= 32) s_error[tidx] += s_error[tidx + 16];
    if(blockSize >= 16) s_error[tidx] += s_error[tidx + 8];
    if(blockSize >= 8) s_error[tidx] += s_error[tidx + 4];
    if(blockSize >= 4) s_error[tidx] += s_error[tidx + 2];
    if(blockSize >= 2) s_error[tidx] += s_error[tidx + 1];
  }

  /* Final result to the output */
  if(tidx == 0) d_error[0] = s_error[0];

}
